#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <string>
#include "./include/util.hpp"
#define THREAD_PER_BLOCK 1024

template <unsigned int blockSize>
__device__ __forceinline__ float warp_shfl_Reduce(float sum) {
    if (blockSize >= 32) sum += __shfl_down_sync(0xffffffff, sum, 16);  // 0-16, 1-17, 2-18, etc.
    if (blockSize >= 16) sum += __shfl_down_sync(0xffffffff, sum, 8);   // 0-8, 1-9, 2-10, etc.
    if (blockSize >= 8) sum += __shfl_down_sync(0xffffffff, sum, 4);    // 0-4, 1-5, 2-6, etc.
    if (blockSize >= 4) sum += __shfl_down_sync(0xffffffff, sum, 2);    // 0-2, 1-3, 4-6, 5-7, etc.
    if (blockSize >= 2) sum += __shfl_down_sync(0xffffffff, sum, 1);    // 0-1, 2-3, 4-5, etc.
    return sum;
}


// template <unsigned int blockSize, int NUM_PER_THREAD>
// __global__ void reduce_v7_shuffle(float *vec_A, float *vec_B) {
//     float *A_start = vec_A + blockIdx.x * blockDim.x * NUM_PER_THREAD;
//     float *B_start = vec_B + blockIdx.x;
//     float sum = 0;
// #pragma unroll 4
//     for (int iter = 0; iter < NUM_PER_THREAD; iter++) {
//         sum += A_start[iter * blockSize + threadIdx.x];
//     }
//     // WARP_SIZE 32
//     __shared__ float warpLevelSums[32];
//     const int laneId = threadIdx.x % 32;
//     const int warpId = threadIdx.x / 32;

//     sum = warp_shfl_Reduce<blockSize>(sum);

//     // 每个 warp 中第一个 thread 存储 warp sum
//     if (laneId == 0) warpLevelSums[warpId] = sum;
//     __syncthreads();

//     // 第一个 warp 再对所有的 warp sum 进行求和
//     // 对第一个 warp sum 重新赋值
//     sum = (threadIdx.x < blockDim.x / 32) ? warpLevelSums[laneId] : 0;
//     if (warpId == 0) sum = warp_shfl_Reduce<blockSize / 32>(sum);
//     if (threadIdx.x == 0) B_start[0] = sum;
// }

template <unsigned int blockSize, int NUM_PER_THREAD>
__global__ void reduce_v7_shuffle(float *vec_A, float *vec_B) {
    float sum = 0;

    // 直接计算，避免存储指针
    // #pragma unroll 
    for (int iter = 0; iter < NUM_PER_THREAD; iter++) {
        sum += vec_A[blockIdx.x * blockDim.x * NUM_PER_THREAD + iter * blockSize + threadIdx.x];
    }

    __shared__ float warpLevelSums[32];

    sum = warp_shfl_Reduce<blockSize>(sum);

    if ((threadIdx.x & 31) == 0) warpLevelSums[threadIdx.x >> 5] = sum;

    __syncthreads();

    if (threadIdx.x < 32) {
        sum = (threadIdx.x < blockDim.x / 32) ? warpLevelSums[threadIdx.x] : 0;
        sum = warp_shfl_Reduce<blockSize / 32>(sum);
    }

    if (threadIdx.x == 0) vec_B[blockIdx.x] = sum;
}

int main(int agrc, char **argv) {
    const int vector_size = 32 * 1024 * 1024;
    const int NUM_PER_THREAD = 4;
    const int BLOCK_NUM = (vector_size + THREAD_PER_BLOCK - 1) / (THREAD_PER_BLOCK * NUM_PER_THREAD);
    float *vector_host = (float *)malloc(vector_size * sizeof(float));
    generateRandomFloatArray(vector_host, vector_size);
    float *vector_device = NULL;
    hipMalloc((void **)&vector_device, vector_size * sizeof(float));
    hipMemcpy(vector_device, vector_host, vector_size * sizeof(float), hipMemcpyHostToDevice);

    float *vector_host_out = (float *)malloc(BLOCK_NUM * sizeof(float));
    float *vector_device_out = NULL;
    hipMalloc((void **)&vector_device_out, BLOCK_NUM * sizeof(float));

    float cpu_result = cpu_reduce(vector_host, vector_size);
    float gpu_result = 0.0;

    dim3 Grid(BLOCK_NUM);
    dim3 Block(THREAD_PER_BLOCK);
    for (int i = 0; i < 5; i++) {
        reduce_v7_shuffle<THREAD_PER_BLOCK, NUM_PER_THREAD><<<Grid, Block>>>(vector_device, vector_device_out);
        hipDeviceSynchronize();
    }
    // reduce_v7_shuffle<THREAD_PER_BLOCK, NUM_PER_THREAD><<<Grid, Block>>>(vector_device, vector_device_out);
    // hipDeviceSynchronize();
    hipMemcpy(vector_host_out, vector_device_out, BLOCK_NUM * sizeof(float), hipMemcpyDeviceToHost);
    gpu_result = cpu_reduce(vector_host_out, BLOCK_NUM);

    std::cout << "cpu result: " << cpu_result << std::endl;
    std::cout << "gpu result: " << gpu_result << std::endl;

    compare_matrices(cpu_result, gpu_result);
    hipFree(vector_device);
    hipFree(vector_device_out);
    return 0;
}