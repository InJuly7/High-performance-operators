#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <string>

#define THREAD_PER_BLOCK 256
static int checkCudaError(hipError_t code, const char *expr, const char *file, int line) {
    if (code) {
        printf("CUDA error at %s:%d, code=%d (%s) in '%s'\n", file, line, (int)code, hipGetErrorString(code), expr);
        return 1;
    }
    return 0;
}

#define checkCudaErr(...)                                                        \
    do {                                                                         \
        int err = checkCudaError(__VA_ARGS__, #__VA_ARGS__, __FILE__, __LINE__); \
    } while (0)

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    // 偶数号block的首地址
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;
    // unrolling 2 data blocks
    if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];
    __syncthreads();
    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[ti] += idata[tid + stride];
        }d
        // synchronize within threadblock
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrollWarps8(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)  // 注意此处的stride循环条件发生了变化(相较于上面的函数)
    {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
    /*
    上面这段代码可以看成
    for (int stride = 32; stride > 0; stride >>= 1)// 注意此处的stride循环条件发生了变化(相较于上面的函数)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }
    实际上就是把上面的循环展开了
    */
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}