#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <string>
#include "./include/util.hpp"

#define WARP_SIZE 32
#define HALF2(val) (reinterpret_cast<half2 *>(&(val)))[0]

using half_t = half_float::half;

__device__ __forceinline__ half warp_reduce_sum_f16(half val) {
#pragma unroll
    for (int mask = WARP_SIZE >> 1; mask >= 1; mask >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, mask);
    }
    return val;
}

template <unsigned int NUM_THREADS>
__device__ __forceinline__ half block_reduce_sum_f16(half val) {
    const int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int laneId = threadIdx.x & (WARP_SIZE - 1);
    static __shared__ float warpsum[NUM_WARPS];
    val = warp_reduce_sum_f16(val);
    if (laneId == 0) warpsum[warpId] = val;
    __syncthreads();
    // tid == 0 返回 block_reduce_sum
    if (warpId == 0) {
        val = (laneId < NUM_WARPS) ? warpsum[laneId] : 0.0f;
        val = warp_reduce_sum_f16(val);
    }
    return val;
}

__device__ __forceinline__ half warp_reduce_max_f16(half val) {
#pragma unroll
    for (int mask = WARP_SIZE >> 1; mask >= 1; mask >>= 1) {
        val = __hmax(val, __shfl_down_sync(0xffffffff, val, mask));
    }
    return val;
}

template <unsigned int NUM_THREADS>
__device__ __forceinline__ half block_reduce_max_f16(half val) {
    const int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int laneId = threadIdx.x & (WARP_SIZE - 1);
    static __shared__ half warpsum[NUM_WARPS];
    val = warp_reduce_max_f16(val);
    if (laneId == 0) warpsum[warpId] = val;
    __syncthreads();
    // tid == 0 返回 block_reduce_max
    if (warpId == 0) {
        val = (laneId < NUM_WARPS) ? warpsum[laneId] : (half)0.0f;
        val = warp_reduce_max_f16(val);
    }
    return val;
}

// NOTE: softmax per-token
// Softmax x: (S,h), y: (S,h)
// grid(S*h/h), block(h), assume h<=1024
// one token per thread block, only support 64<=h<=1024 and 2^n
// HEAD_SIZE/KV_LEN=NUM_THREADS ??? 没看懂,
// e^x_i/sum(e^x_0,...,e^x_n-1)
#define HALF2MAX(reg_x, reg_y) __hmax((reg_x), (reg_y))
#define HALF4MAX(reg_x, reg_y, reg_z, reg_w) __hmax(HALF2MAX(reg_x, reg_y), HALF2MAX(reg_z, reg_w))
#define HALF2_EXP(reg, global_max, local_sum) \
    (reg).x = hexp((reg).x - global_max);     \
    (reg).y = hexp((reg).y - global_max);     \
    local_sum += (reg).x;                     \
    local_sum += (reg).y;
#define HALF2_SOFTMAX(reg_B, reg_A, global_sum) \
    (reg_B).x = __hdiv((reg_A).x, global_sum);  \
    (reg_B).y = __hdiv((reg_A).y, global_sum);

template <unsigned int NUM_THREADS>
__global__ void safe_softmax_v3_f16x8_f16(half *mat_A, half *mat_B, int N) {
    half *thread_A_start = mat_A + blockIdx.x * N + 8 * threadIdx.x;
    half *thread_B_start = mat_B + blockIdx.x * N + 8 * threadIdx.x;

    __shared__ half exp_sum;
    __shared__ half global_max;

    half local_max = __float2half(-65504.0f);
    half2 reg_A_0 = HALF2(thread_A_start[0]);
    half2 reg_A_1 = HALF2(thread_A_start[2]);
    half2 reg_A_2 = HALF2(thread_A_start[4]);
    half2 reg_A_3 = HALF2(thread_A_start[6]);

    local_max = HALF2MAX(HALF4MAX(reg_A_0.x, reg_A_0.y, reg_A_1.x, reg_A_1.y), HALF4MAX(reg_A_2.x, reg_A_2.y, reg_A_3.x, reg_A_3.y));
    local_max = block_reduce_max_f16<NUM_THREADS>(local_max);
    if (threadIdx.x == 0) global_max = local_max;
    __syncthreads();

    half local_sum = __float2half(0.0f);
    HALF2_EXP(reg_A_0, global_max, local_sum);
    HALF2_EXP(reg_A_1, global_max, local_sum);
    HALF2_EXP(reg_A_2, global_max, local_sum);
    HALF2_EXP(reg_A_3, global_max, local_sum);
    local_sum = block_reduce_sum_f16<NUM_THREADS>(local_sum);
    if (threadIdx.x == 0) exp_sum = local_sum;
    __syncthreads();

    half2 reg_B_0, reg_B_1, reg_B_2, reg_B_3;
    HALF2_SOFTMAX(reg_B_0, reg_A_0, exp_sum);
    HALF2_SOFTMAX(reg_B_1, reg_A_1, exp_sum);
    HALF2_SOFTMAX(reg_B_2, reg_A_2, exp_sum);
    HALF2_SOFTMAX(reg_B_3, reg_A_3, exp_sum);
    HALF2(thread_B_start[0]) = reg_B_0;
    HALF2(thread_B_start[2]) = reg_B_1;
    HALF2(thread_B_start[4]) = reg_B_2;
    HALF2(thread_B_start[6]) = reg_B_3;
}

int main() {
    const int N1 = 4096;
    const int N2 = 1024;
    half_t *mat_A = (half_t *)malloc(N1 * N2 * sizeof(half_t));
    half_t *mat_B_cpu_calc = (half_t *)malloc(N1 * N2 * sizeof(half_t));
    generateRandomHalfArray(mat_A, N1 * N2);
    half *mat_A_device = NULL;
    hipMalloc((void **)&mat_A_device, N1 * N2 * sizeof(half));
    hipMemcpy(mat_A_device, mat_A, N1 * N2 * sizeof(half), hipMemcpyHostToDevice);

    cpu_safe_softmax(mat_A, mat_B_cpu_calc, N1, N2);

    half *mat_B_device = NULL;
    half_t *mat_B_gpu_calc = (half_t *)malloc(N1 * N2 * sizeof(half_t));
    hipMalloc((void **)&mat_B_device, N1 * N2 * sizeof(half));
    dim3 grid(N1);
    dim3 block(N2 / 8);

    for (int i = 0; i < 5; i++) {
        Perf perf("safe_softmax_v3_f16x8_f16");
        safe_softmax_v3_f16x8_f16<N2 / 8><<<grid, block>>>(mat_A_device, mat_B_device, N2);
    }

    hipMemcpy(mat_B_gpu_calc, mat_B_device, N1 * N2 * sizeof(half), hipMemcpyDeviceToHost);
    printHalfArray(mat_B_cpu_calc, 10);
    printHalfArray(mat_B_gpu_calc, 10);
    compare_matrices(N1, N2, mat_B_cpu_calc, mat_B_gpu_calc);

    free(mat_A);
    free(mat_B_cpu_calc);
    free(mat_B_gpu_calc);
    hipFree(mat_A_device);
    hipFree(mat_B_device);
}