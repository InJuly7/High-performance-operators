#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <iostream>

#include <random>
#include <string>
#include "./include/util.hpp"

#define WARP_SIZE 32

#define LDST128BITS(val) (reinterpret_cast<float4 *>(&(val)))[0]

using half_t = half_float::half;

__device__ __forceinline__ half warp_reduce_sum_f16(half val) {
#pragma unroll
    for (int mask = WARP_SIZE >> 1; mask >= 1; mask >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, mask);
    }
    return val;
}

template <unsigned int NUM_THREADS>
__device__ __forceinline__ half block_reduce_sum_f16(half val) {
    const int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int laneId = threadIdx.x & (WARP_SIZE - 1);
    static __shared__ float warpsum[NUM_WARPS];
    val = warp_reduce_sum_f16(val);
    if (laneId == 0) warpsum[warpId] = val;
    __syncthreads();
    // tid == 0 返回 block_reduce_sum 
    if (warpId == 0) {
        val = (laneId < NUM_WARPS) ? warpsum[laneId] : 0.0f;
        val = warp_reduce_sum_f16(val);
    }
    return val;
}

__device__ __forceinline__ half warp_reduce_max_f16(half val) {
#pragma unroll
    for (int mask = WARP_SIZE >> 1; mask >= 1; mask >>= 1) {
        val = __hmax(val, __shfl_down_sync(0xffffffff, val, mask));
    }
    return val;
}

template <unsigned int NUM_THREADS>
__device__ __forceinline__ half block_reduce_max_f16(half val) {
    const int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int laneId = threadIdx.x & (WARP_SIZE - 1);
    static __shared__ half warpsum[NUM_WARPS];
    val = warp_reduce_max_f16(val);
    if (laneId == 0) warpsum[warpId] = val;
    __syncthreads();
    // tid == 0 返回 block_reduce_max 
    if (warpId == 0) {
        val = (laneId < NUM_WARPS) ? warpsum[laneId] : (half)0.0f;
        val = warp_reduce_max_f16(val);
    }
    return val;
}


// NOTE: softmax per-token
// Softmax x: (S,h), y: (S,h)
// grid(S*h/h), block(h), assume h<=1024
// one token per thread block, only support 64<=h<=1024 and 2^n
// HEAD_SIZE/KV_LEN=NUM_THREADS ??? 没看懂,
// e^x_i/sum(e^x_0,...,e^x_n-1)
template <unsigned int NUM_THREADS>
__global__ void safe_softmax_v2_f16x8_pack_f16(half *mat_A, half *mat_B, int N) {
    half *thread_A_start = mat_A + blockIdx.x * N + 8 * threadIdx.x;
    half *thread_B_start = mat_B + blockIdx.x * N + 8 * threadIdx.x;
    half pack_A[8], pack_B[8];
    LDST128BITS(pack_A[0]) = LDST128BITS(thread_A_start[0]);
    __shared__ half exp_sum;
    __shared__ half global_max;

    half local_max = __float2half(-65504.0f);
    for(int i = 0; i < 8; i++) {
        local_max = __hmax(local_max, pack_A[i]);
    }
    local_max = block_reduce_max_f16<NUM_THREADS>(local_max);
    if(threadIdx.x == 0) global_max = local_max;
    __syncthreads();

    half local_sum = __float2half(0.0f);
    for(int i = 0; i < 8; i++) {
        pack_A[i] = hexp(pack_A[i] - global_max);
        local_sum += pack_A[i];
    }
    local_sum = block_reduce_sum_f16<NUM_THREADS>(local_sum);
    if(threadIdx.x == 0) exp_sum = local_sum;
    __syncthreads();
    
    for(int i = 0; i < 8; i++) {
        pack_B[i] = pack_A[i]/exp_sum;
    }
    LDST128BITS(thread_B_start[0]) = LDST128BITS(pack_B[0]);
}

int main() {
    const int N1 = 4096;
    const int N2 = 1024;
    half_t *mat_A = (half_t *)malloc(N1 * N2 * sizeof(half_t));
    half_t *mat_B_cpu_calc = (half_t *)malloc(N1 * N2 * sizeof(half_t));
    generateRandomHalfArray(mat_A, N1 * N2);
    half *mat_A_device = NULL;
    hipMalloc((void **)&mat_A_device, N1 * N2 * sizeof(half));
    hipMemcpy(mat_A_device, mat_A, N1 * N2 * sizeof(half), hipMemcpyHostToDevice);

    cpu_safe_softmax(mat_A, mat_B_cpu_calc, N1, N2);

    half *mat_B_device = NULL;
    half_t *mat_B_gpu_calc = (half_t *)malloc(N1 * N2 * sizeof(half_t));
    hipMalloc((void **)&mat_B_device, N1 * N2 * sizeof(half));
    dim3 grid(N1);
    dim3 block(N2/8);

    for (int i = 0; i < 5; i++) {
        Perf perf("safe_softmax_v2_f16x8_pack_f16");
        safe_softmax_v2_f16x8_pack_f16<N2/8><<<grid, block>>>(mat_A_device, mat_B_device, N2);
    }

    hipMemcpy(mat_B_gpu_calc, mat_B_device, N1 * N2 * sizeof(half), hipMemcpyDeviceToHost);
    printHalfArray(mat_B_cpu_calc, 10);
    printHalfArray(mat_B_gpu_calc, 10);
    compare_matrices(N1, N2, mat_B_cpu_calc, mat_B_gpu_calc);

    free(mat_A);
    free(mat_B_cpu_calc);
    free(mat_B_gpu_calc);
    hipFree(mat_A_device);
    hipFree(mat_B_device);
}