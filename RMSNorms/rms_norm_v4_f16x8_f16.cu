#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>
#include <random>
#include <string>

#include "./include/util.hpp"
#include "/home/song/program/High-performance-operators/include/half.hpp"

using half_t = half_float::half;

#define WARP_SIZE 32
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])

__device__ __forceinline__ half warp_reduce_sum_f16_f16(half val) {
#pragma unroll
    for (int mask = WARP_SIZE >> 1; mask >= 1; mask >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, mask);
    }
    return val;
}

template <unsigned int NUM_THREADS>
__device__ __forceinline__ half block_reduce_sum_f16_f16(half val) {
    const int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int laneId = threadIdx.x & (WARP_SIZE - 1);
    static __shared__ half warpsum[NUM_WARPS];
    val = warp_reduce_sum_f16_f16(val);
    if (laneId == 0) warpsum[warpId] = val;
    __syncthreads();
    if (warpId == 0) {
        val = (laneId < NUM_WARPS) ? warpsum[laneId] : __float2half(0.0f);
        val = warp_reduce_sum_f16_f16(val);
    }
    return val;
}

// RMS Norm: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
#define HALF2_VARIANCE(reg) (reg).x *(reg).x + (reg).y *(reg).y
#define HALF2_RMS_NORM(reg_y, reg_x, s_variance, g)           \
    do {                                          \
        (reg_y).x = (reg_x).x * s_variance * (g); \
        (reg_y).y = (reg_x).y * s_variance * (g); \
    } while (0)

template <unsigned int NUM_THREADS>
__global__ void rms_norm_v4_f16x8_f16(half *mat_A, half *mat_B, float g, int N, int K) {
    half *thread_A_start = mat_A + blockIdx.x * K + threadIdx.x * 8;
    half *thread_B_start = mat_B + blockIdx.x * K + threadIdx.x * 8;
    const half epsilon = __float2half(1e-5f);
    const half g_ = __float2half(g);
    const half K_ = __int2half_rn(K);
    // 块内共享, 求出当前行 rsqrtf(sum(ai^2)/K)
    __shared__ half s_variance;

    half2 reg_A_0 = HALF2(thread_A_start[0]);
    half2 reg_A_1 = HALF2(thread_A_start[2]);
    half2 reg_A_2 = HALF2(thread_A_start[4]);
    half2 reg_A_3 = HALF2(thread_A_start[6]);

    half variance = HALF2_VARIANCE(reg_A_0);
    variance += HALF2_VARIANCE(reg_A_1);
    variance += HALF2_VARIANCE(reg_A_2);
    variance += HALF2_VARIANCE(reg_A_3);

    variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
    if (threadIdx.x == 0) s_variance = hrsqrt(variance / K_ + epsilon);
    __syncthreads();
    half2 reg_B_0, reg_B_1,reg_B_2,reg_B_3;
    HALF2_RMS_NORM(reg_B_0, reg_A_0, s_variance, g_);
    HALF2_RMS_NORM(reg_B_1, reg_A_1, s_variance, g_);
    HALF2_RMS_NORM(reg_B_2, reg_A_2, s_variance, g_);
    HALF2_RMS_NORM(reg_B_3, reg_A_3, s_variance, g_);
    HALF2(thread_B_start[0]) = reg_B_0;
    HALF2(thread_B_start[2]) = reg_B_1;
    HALF2(thread_B_start[4]) = reg_B_2;
    HALF2(thread_B_start[6]) = reg_B_3;
}

int main() {
    const int N = 4096;
    const int K = 1024;
    float g = 0.35f;

    // CPU 内存分配 - 都使用 half_t
    half_t *mat_A = (half_t *)malloc(N * K * sizeof(half_t));
    half_t *mat_B_cpu_calc = (half_t *)malloc(N * K * sizeof(half_t));
    half_t *mat_B_gpu_calc = (half_t *)malloc(N * K * sizeof(half_t));

    generateRandomHalfArray(mat_A, N * K);

    // GPU 内存分配 - 都使用 half
    half *mat_A_device = NULL;
    half *mat_B_device = NULL;
    hipMalloc((void **)&mat_A_device, N * K * sizeof(half_t));
    hipMalloc((void **)&mat_B_device, N * K * sizeof(half_t));
    hipMemcpy(mat_A_device, mat_A, N * K * sizeof(half_t), hipMemcpyHostToDevice);
    cpu_rms_norm(mat_A, mat_B_cpu_calc, g, N, K);

    dim3 grid(N);
    dim3 block(K/8);
    for (int i = 0; i < 5; i++) {
        Perf perf("rms_norm_v4_f16x8_f16");
        rms_norm_v4_f16x8_f16<K/8><<<grid, block>>>(mat_A_device, mat_B_device, g, N, K);
    }
    hipMemcpy(mat_B_gpu_calc, mat_B_device, N * K * sizeof(half_t), hipMemcpyDeviceToHost);

    printHalfArray(mat_B_cpu_calc, 10);
    printHalfArray(mat_B_gpu_calc, 10);
    compare_matrices(N, K, mat_B_cpu_calc, mat_B_gpu_calc);

    free(mat_A);
    free(mat_B_cpu_calc);
    free(mat_B_gpu_calc);
    hipFree(mat_A_device);
    hipFree(mat_B_device);

    return 0;
}