#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "./include/util.hpp"
#include "../cublas/cublas.cuh"
#include "../include/cuda_log.cuh"

using namespace nvcuda;
using half_t = half_float::half;

#define WARP_SIZE 32
#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

// Vector Access
#define HALF2(value) (reinterpret_cast<half2 *>(&(value)))[0]
#define HALF4(value) (reinterpret_cast<float2 *>(&(value)))[0]
#define HALF8(value) (reinterpret_cast<float4 *>(&(value)))[0]
#define FLOAT2(val) (reinterpret_cast<float2 *>(&(value)))[0]
#define FLOAT4(val) (reinterpret_cast<float4 *>(&(val)))[0]
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value)))[0]
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value)))[0]
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value)))[0]
#define REG(val) (*reinterpret_cast<uint32_t *>(&(val)))

// PTX ISA
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))

#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) \
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr)                                 \
    asm volatile(                                                           \
        "ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, " \
        "[%4];\n"                                                           \
        : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                            \
        : "r"(addr))

#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1)             \
    asm volatile(                                                               \
        "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, " \
        "%4, %5}, {%6, %7}, {%8, %9};\n"                                        \
        : "=r"(RD0), "=r"(RD1)                                                  \
        : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

template <uint32_t shift, uint32_t xor_bits, uint32_t chunk_bits>
__device__ __forceinline__ uint32_t swizzle(const uint32_t offset, const uint32_t SMem_col) {
    const int BMask = ((1 << xor_bits) - 1) << chunk_bits;
    const int swizzle_offset = ((offset >> shift) & BMask) ^ offset;
    const int swizzle_col = (swizzle_offset & (SMem_col - 1));
    return swizzle_col;
}

template <unsigned int MMA_M, unsigned int MMA_K, unsigned int MMA_N>
__global__ void hgemmT_v0_mma_m16n8k16_W1x1_T1x1(half *A, half *B, half *C, const int M, const int K, const int N) {
    const int BM = MMA_M;
    const int BK = MMA_K;
    const int BN = MMA_N;

    A += blockIdx.y * BM * K;
    B += blockIdx.x * BN * K;
    C += blockIdx.y * BM * N + blockIdx.x * BN;

    __shared__ half SMem_A[BM][BK];
    __shared__ half SMem_B[BN][BK];
    uint32_t RA[4], RB[2];
    uint32_t RC[2] = {0,0};

    int warpId = threadIdx.x / WARP_SIZE;
    int laneId = threadIdx.x & (WARP_SIZE - 1);
    
    // (16 * 16) / 32 = 8 E/T
    int LD_GMemA_Row = (threadIdx.x * 8) / 16;
    int LD_GMemA_Col = (threadIdx.x * 8) & 15;

    // (8 * 16) / 32 = 4 E/T
    int LD_GMemB_Row = (threadIdx.x * 4) / 16;
    int LD_GMemB_Col = (threadIdx.x * 4) & 15;

    // fp16 SMem[16][16]
    const int row_bits = 4;
    const int col_bits = 4;
    const int chunk_col_bits = 1;
    const int chunk_bits = 3; // M
    const int stride_bits = 3; // half 8
    const int mma_row_bits = 3; // 8 rows
    const int bank_bits = 5; // 32 banks
    // stride_bits = 4 ==> 2, stride_bits = 3 ==> 1
    // https://zhuanlan.zhihu.com/p/21142007017
    const int xor_bits = (stride_bits >= 5 ) ? 3 : stride_bits + mma_row_bits - bank_bits;
    const int shift = (stride_bits >= 5) ? stride_bits - 2 : 3;  // S

    for (int k = 0; k < K; k += BK) {
        // Load GMemA/B  Store SMemA/B
        int offset = LD_GMemA_Row * BK + LD_GMemA_Col;
        // swizzle<3,1,3>
        int swizzle_col = swizzle<shift, xor_bits, chunk_bits>(offset, BK);
        HALF8(SMem_A[LD_GMemA_Row][swizzle_col]) = HALF8(A[LD_GMemA_Row * K + LD_GMemA_Col]);
        HALF4(SMem_B[LD_GMemB_Row][LD_GMemB_Col]) = HALF4(B[LD_GMemB_Row * K + LD_GMemB_Col]);
        A += BK;
        B += BK;
        // cudaLog("row : %d, swizzle_col : %d\n",LD_GMemA_Row, swizzle_col);
        __syncthreads();
    
        // Load SMemA/B  Store RegA/B
        // x4.m8n8
        int RegA_Ptr_Row = laneId & 15;
        int RegA_Ptr_Col = (laneId / 16) * 8;
        offset = RegA_Ptr_Row * BK + RegA_Ptr_Col;
        swizzle_col = swizzle<shift, xor_bits, chunk_bits>(offset, BK);
        uint32_t LD_SMemA_Ptr = __cvta_generic_to_shared(&SMem_A[RegA_Ptr_Row][swizzle_col]);

        // x2.m8n8
        int RegB_Ptr_Row = laneId & 7;
        int RegB_Ptr_Col = (laneId / 8) * 8;
        uint32_t LD_SMemB_Ptr = __cvta_generic_to_shared(&SMem_B[RegB_Ptr_Row][RegB_Ptr_Col]);

        // for(int i = 0; i < 4; i++) {
        //     int Row_offset, Col_offset;
        //     if (i == 0) {
        //         // T0~T3 : T0, ... , T28~T31 : T7
        //         Row_offset = (laneId / 4) & 15, Col_offset = ((laneId / 4) / 16) * 8 + ((laneId * 2) & 7);  // T0~T7 控制
        //     } else if (i == 1) {
        //         // T0~T3 : T8, ... , T28~T31 : T15
        //         Row_offset = (laneId / 4 + 8) & 15, Col_offset = ((laneId / 4 + 8) / 16) * 8 + ((laneId * 2) & 7);  // T8~T15 控制
        //     } else if (i == 2) {
        //         // T0~T3 : T16, ... , T28~T31 : T23
        //         Row_offset = (laneId / 4 + 16) & 15, Col_offset = ((laneId / 4 + 16) / 16) * 8 + ((laneId * 2) & 7);  // T16~T23 控制
        //     } else if (i == 3) {
        //         // T0~T3 : T24, ... , T28~T31 : T31
        //         Row_offset = (laneId / 4 + 24) & 15, Col_offset = ((laneId / 4 + 24) / 16) * 8 + ((laneId * 2) & 7);  // T24~T31 控制
        //     }

        //     int bankId = ((Row_offset * BK + Col_offset) / 2) & 31;
        //     cudaLog("RA[%d]: (%d, %d) bankId: %d\n", i, Row_offset, Col_offset, bankId);
        // }

        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], LD_SMemA_Ptr);
        LDMATRIX_X2(RB[0], RB[1], LD_SMemB_Ptr);
        // for(int i = 0; i < 4; i++) {
        //     float2 temp;
        //     temp = __half22float2(HALF2(RA[i]));
        //     cudaLog("RA[%d]: (%d, %d) (%f,%f)\n", i, RegA_Ptr_Row, RegA_Ptr_Col, temp.x, temp.y);
        // }

        // MMA
        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);
        __syncthreads();
    }

    // Store RegC  Store GMemC
    int ST_GMemC_Row = (threadIdx.x * 2) / 8;
    int ST_GMemC_Col = (threadIdx.x * 2) & 7;
    LDST32BITS(C[ST_GMemC_Row * N + ST_GMemC_Col]) = LDST32BITS(RC[0]);
    LDST32BITS(C[(ST_GMemC_Row + 8) * N + ST_GMemC_Col]) = LDST32BITS(RC[1]);
}

int main() {
    const int M = 16;
    const int K = 16;
    const int N = 8;

    half_t *A = (half_t *)malloc(M * K * sizeof(half_t));
    half_t *B = (half_t *)malloc(N * K * sizeof(half_t));
    half_t *C_cublas_cal = (half_t *)malloc(M * N * sizeof(half_t));
    half_t *C_mma_cal = (half_t *)malloc(M * N * sizeof(half_t));

    generateRandomHalfArray(A, M * K, true, "A.txt");
    generateRandomHalfArray(B, N * K, true, "B.txt");

    // d_B N * K
    half *d_A, *d_B, *d_C_mma, *d_C_cublas;
    hipMalloc((void **)&d_A, M * K * sizeof(half));
    hipMalloc((void **)&d_B, N * K * sizeof(half));
    hipMalloc((void **)&d_C_mma, M * N * sizeof(half));
    hipMalloc((void **)&d_C_cublas, M * N * sizeof(half));

    hipMemcpy(d_A, A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * K * sizeof(half), hipMemcpyHostToDevice);

    // d_B N * K
    hgemmT_cublas(d_A, d_B, d_C_cublas, M, K, N);
    hipMemcpy(C_cublas_cal, d_C_cublas, M * N * sizeof(half), hipMemcpyDeviceToHost);

    const int BM = 16;
    const int BK = 16;
    const int BN = 8;
    dim3 grid(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 block(32);
    hgemmT_v0_mma_m16n8k16_W1x1_T1x1<BM, BK, BN><<<grid, block>>>(d_A, d_B, d_C_mma, M, K, N);
    hipMemcpy(C_mma_cal, d_C_mma, M * N * sizeof(half), hipMemcpyDeviceToHost);
    printHalfArray(C_cublas_cal, 10);
    printHalfArray(C_mma_cal, 10);
    compare_matrices(M, N, C_cublas_cal, C_mma_cal);

    free(A);
    free(B);
    free(C_cublas_cal);
    free(C_mma_cal);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_mma);
    hipFree(d_C_cublas);
    return 0;
}

