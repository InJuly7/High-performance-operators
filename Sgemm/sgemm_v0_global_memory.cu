#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "./include/util.hpp"

// 每个Thread读2K元素, 写入1个元素
__global__ void cuda_sgemm(float *matrix_A_device, float *matrix_B_device, float *matrix_C_device, int M, int N, int K) {
    float *A_ptr_start = matrix_A_device + blockDim.y * blockIdx.y * K;
    float *B_ptr_start = matrix_B_device + blockDim.x * blockIdx.x;
    float *C_ptr_start = matrix_C_device + blockDim.y * blockIdx.y * N + blockDim.x * blockIdx.x;
    float temp = 0.0f;
    for (int k = 0; k < K; k++) {
        temp += A_ptr_start[threadIdx.y * K + k] * B_ptr_start[k * N + threadIdx.x];
    }
    C_ptr_start[threadIdx.y * N + threadIdx.x] = temp;
}

int main() {
    int m = 256, n = 256, k = 256;
    const int mem_size_A = m * k * sizeof(float);
    const int mem_size_B = k * n * sizeof(float);
    const int mem_size_C = m * n * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    generateRandomFloatArray(matrix_A_host, m * k);
    generateRandomFloatArray(matrix_B_host, k * n);

    memset(matrix_C_host_cpu_calc, 0, mem_size_C);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

    printFloatArray(matrix_C_host_cpu_calc, 10);

    const int BLOCK = 16;
    dim3 block(BLOCK, BLOCK);
    dim3 grid((n + BLOCK - 1) / BLOCK, (m + BLOCK - 1) / BLOCK);
    cuda_sgemm<<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);
    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);
    printFloatArray(matrix_C_host_gpu_calc, 10);

    compare_matrices(m * n, matrix_C_host_cpu_calc, matrix_C_host_gpu_calc);

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
}
