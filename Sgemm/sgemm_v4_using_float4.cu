#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "./include/util.hpp"

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
template <unsigned int BM, unsigned int BK, unsigned int BN, unsigned int NUM_PER_THREAD, unsigned int M, unsigned int K, unsigned int N>
__global__ void cuda_sgemm(float *matrix_A_device, float *matrix_B_device, float *matrix_C_device) {
    float *A_ptr_start = matrix_A_device + blockIdx.y * BM * K;
    float *B_ptr_start = matrix_B_device + blockIdx.x * BN;
    float *C_ptr_start = matrix_C_device + blockIdx.y * BM * N + blockIdx.x * BN;

    __shared__ float A_smem[BM][BK];
    __shared__ float B_smem[BK][BN];

    float temp[NUM_PER_THREAD] = {0.0f};

    for (int i = 0; i < K; i += BK) {
        // GM ==> SM
        // 将相同内存地址的数据重新解释为一个 float4 对象
        FETCH_FLOAT4(A_smem[threadIdx.y][threadIdx.x * NUM_PER_THREAD]) =
            FETCH_FLOAT4(A_ptr_start[threadIdx.y * K + threadIdx.x * NUM_PER_THREAD + i]);
        FETCH_FLOAT4(B_smem[threadIdx.y][threadIdx.x * NUM_PER_THREAD]) =
            FETCH_FLOAT4(B_ptr_start[(i + threadIdx.y) * N + threadIdx.x * NUM_PER_THREAD]);

        __syncthreads();

        for (int j = 0; j < NUM_PER_THREAD; j++) {
            for (int k = 0; k < BK; k++) {
                temp[j] += A_smem[threadIdx.y][k] * B_smem[k][threadIdx.x * NUM_PER_THREAD + j];
            }
        }
        __syncthreads();
    }

    for (int i = 0; i < NUM_PER_THREAD; i++) {
        C_ptr_start[threadIdx.y * N + threadIdx.x * NUM_PER_THREAD + i] = temp[i];
    }
}

int main() {
    const int m = 256, k = 256, n = 256;
    const int mem_size_A = m * k * sizeof(float);
    const int mem_size_B = k * n * sizeof(float);
    const int mem_size_C = m * n * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    generateRandomFloatArray(matrix_A_host, m * k);
    generateRandomFloatArray(matrix_B_host, k * n);

    memset(matrix_C_host_cpu_calc, 0, mem_size_C);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

    printFloatArray(matrix_C_host_cpu_calc, 10);

    const int bm = 32;
    const int bk = 32;
    const int bn = 32;
    const int BLOCK_X = 8;
    const int BLOCK_Y = 32;
    dim3 block(BLOCK_X, BLOCK_Y);
    dim3 grid(n / bn, m / bm);
    cuda_sgemm<bm, bk, bn, bn / BLOCK_X, m, k, n><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);
    printFloatArray(matrix_C_host_gpu_calc, 10);

    compare_matrices(m * n, matrix_C_host_cpu_calc, matrix_C_host_gpu_calc);

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
}
