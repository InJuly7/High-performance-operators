#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "./include/util.hpp"

template <unsigned int M, unsigned K, unsigned int N, unsigned int BM, unsigned BK, unsigned BN>
__global__ void cuda_sgemm(float *matrix_A_device, float *matrix_B_device, float *matrix_C_device) {
    float *A_ptr_start = matrix_A_device + blockDim.y * blockIdx.y * K;
    float *B_ptr_start = matrix_B_device + blockDim.x * blockIdx.x;
    float *C_ptr_start = matrix_C_device + blockDim.y * blockIdx.y * N + blockDim.x * blockIdx.x;

    __shared__ float A_smem[BM][BK];
    __shared__ float B_smem[BK][BN];

    float temp = 0.0f;
    for (int i = 0; i < K; i += BK) {
        A_smem[threadIdx.y][threadIdx.x] = A_ptr_start[threadIdx.y * K + threadIdx.x + i];
        B_smem[threadIdx.y][threadIdx.x] = B_ptr_start[(threadIdx.y + i) * N + threadIdx.x];

        __syncthreads();

        for (int j = 0; j < BK; j++) {
            temp += A_smem[threadIdx.y][j] * B_smem[j][threadIdx.x];
        }
        __syncthreads();
    }

    C_ptr_start[threadIdx.y * N + threadIdx.x] = temp;
}

int main() {
    const int m = 256, n = 256, k = 256;
    const int mem_size_A = m * k * sizeof(float);
    const int mem_size_B = k * n * sizeof(float);
    const int mem_size_C = m * n * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    generateRandomFloatArray(matrix_A_host, m * k);
    generateRandomFloatArray(matrix_B_host, k * n);

    memset(matrix_C_host_cpu_calc, 0, mem_size_C);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

    printFloatArray(matrix_C_host_cpu_calc, 10);

    const int BM = 16, BN = 16;
    const int BK = 16;
    dim3 block(BN, BM);
    dim3 grid((n + BN - 1) / BN, (m + BM - 1) / BM);
    cuda_sgemm<m, k, n, BM, BK, BN><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device);
    hipDeviceSynchronize();
    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);
    printFloatArray(matrix_C_host_gpu_calc, 10);

    compare_matrices(m * n, matrix_C_host_cpu_calc, matrix_C_host_gpu_calc);

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
}
