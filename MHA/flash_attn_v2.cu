#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "./include/util.hpp"

#define WARP_SIZE 32
#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))
#define LDST128BITS(val) (reinterpret_cast<float4 *>(&(val)))[0]
#define FLOAT4(val) (reinterpret_cast<float4 *>(&(val)))[0]

template <unsigned int sram_size>
__global__ void flash_attn_v2(float *Q, float *K, float *V, float *l, float *m, float *O, int S, int dk, int Tc, int Tr, int Bc, int Br) {
    float scale = rsqrtf(dk);
    int tile_size = Bc * dk;
    __shared__ float SMem[sram_size];
    float *SMem_Q = &SMem[0];
    float *SMem_K = &SMem[tile_size];
    float *SMem_V = &SMem[tile_size * 2];
    float *SMem_S = &SMem[tile_size * 3];
    Q += blockIdx.x * S * dk;
    K += blockIdx.x * S * dk;
    V += blockIdx.x * S * dk;
    O += blockIdx.x * S * dk;
    l += blockIdx.x * S;
    m += blockIdx.x * S;

    for (int tc = 0; tc < Tc; tc++) {
        // Load GMemK, GMemV To SMemK, SMemV
        for (int i = 0; i < dk; i += 4) {
            FLOAT4(SMem_K[threadIdx.x * dk + i]) = FLOAT4(K[tc * tile_size + threadIdx.x * dk + i]);
            FLOAT4(SMem_V[threadIdx.x * dk + i]) = FLOAT4(V[tc * tile_size + threadIdx.x * dk + i]);
        }
        __syncthreads();

        for (int tr = 0; tr < Tr; tr++) {
            int row_idx = tr * Br + threadIdx.x;
            float row_m_prev = m[row_idx];
            float row_l_prev = l[row_idx];
            // Load GMemQ To SMemQ
            for (int i = 0; i < dk; i += 4) {
                FLOAT4(SMem_Q[threadIdx.x * dk + i]) = FLOAT4(Q[row_idx * dk + i]);
            }

            // Q @ K^T
            float row_m = -FLT_MAX;
            float row_l = 0.0f;
            for (int bc = 0; bc < Bc; bc++) {
                float temp = 0.0f;
                for (int i = 0; i < dk; i++) {
                    temp += SMem_Q[threadIdx.x * dk + i] * SMem_K[bc * dk + i];
                }
                SMem_S[threadIdx.x * Bc + bc] = temp * scale;
                // compute row_max
                row_m = fmax(row_m, temp);
            }

            // compute row_l
            for (int bc = 0; bc < Bc; bc++) {
                SMem_S[threadIdx.x * Bc + bc] = __expf(SMem_S[threadIdx.x * Bc + bc] - row_m);
                row_l += SMem_S[threadIdx.x * Bc + bc];
            }

            // update row_m row_l
            float row_m_new = fmax(row_m, row_m_prev);
            float row_l_new = __expf(row_m_prev - row_m_new) * row_l_prev + __expf(row_m - row_m_new) * row_l;

            for (int i = 0; i < dk; i++) {
                float pv = 0.0f;
                for (int bc = 0; bc < Bc; bc++) {
                    pv += SMem_S[threadIdx.x * Bc + bc] * SMem_V[bc * dk + i];
                }
                O[row_idx * dk + i] =
                    (O[row_idx * dk + i]) * __expf(row_m_prev - row_m_new) * (row_l_prev / row_l_new) + (pv * __expf(row_m - row_m_new) / row_l_new);
            }

            m[row_idx] = row_m_new;
            l[row_idx] = row_l_new;
        }
        __syncthreads();
    }
}

int main() {
    const int S1 = 1024;
    const int S2 = 1024;
    const int dk = 64;
    const int H = 12;
    const int Bc = 32;
    const int Br = 32;
    const int Tc = CEIL_DIV(S2, Bc);
    const int Tr = CEIL_DIV(S1, Br);
    const int 
    // Q, K, V tile, S tile
    const int sram_size = (Br * dk + 2 * Bc * dk + Br * Bc);

    float *Q = (float *)malloc(H * S1 * dk * sizeof(float));
    float *K = (float *)malloc(H * S2 * dk * sizeof(float));
    float *V = (float *)malloc(H * S2 * dk * sizeof(float));
    float *m = (float *)malloc(H * S1 * sizeof(float));
    float *l = (float *)malloc(H * S1 * sizeof(float));
    float *O_gpu_cal = (float *)malloc(H * S1 * dk * sizeof(float));
    float *O_cpu_cal = (float *)malloc(H * S1 * dk * sizeof(float));

    generateRandomFloatArray(Q, H * S1 * dk);
    generateRandomFloatArray(K, H * S2 * dk);
    generateRandomFloatArray(V, H * S2 * dk);

    float *d_Q, *d_K, *d_V, *d_m, *d_l, *d_O;
    hipMalloc((void **)&d_Q, H * S1 * dk * sizeof(float));
    hipMalloc((void **)&d_K, H * S2 * dk * sizeof(float));
    hipMalloc((void **)&d_V, H * S2 * dk * sizeof(float));
    hipMalloc((void **)&d_m, H * S1 * sizeof(float));
    hipMalloc((void **)&d_l, H * S1 * sizeof(float));
    hipMalloc((void **)&d_O, H * S1 * dk * sizeof(float));

    hipMemset(d_O, 0, H * S1 * dk * sizeof(float));
    hipMemset(d_m, 0, H * S1 * sizeof(float));
    hipMemset(d_l, 0, H * S1 * sizeof(float));

    hipMemcpy(d_Q, Q, H * S1 * dk * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_K, K, H * S2 * dk * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, H * S2 * dk * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(H);
    dim3 block(Bc);
    flash_attn_v2<sram_size><<<grid, block>>>(d_Q, d_K, d_V, d_l, d_m, d_O, S1, dk, Tc, Tr, Bc, Br);
    hipDeviceSynchronize();
    hipMemcpy(O_gpu_cal, d_O, H * S1 * dk * sizeof(float), hipMemcpyDeviceToHost);

    cpu_multihead_attention(Q, K, V, O_cpu_cal, H, S1, S2, dk);
    printFloatArray(O_cpu_cal, 10);
    printFloatArray(O_gpu_cal, 10);
    compare_matrices(H, S1, dk, O_cpu_cal, O_gpu_cal);

    free(Q);
    free(K);
    free(V);
    free(m);
    free(l);
    free(O_gpu_cal);
    free(O_cpu_cal);

    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_O);
    hipFree(d_l);
    hipFree(d_m);
}